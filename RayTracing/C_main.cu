#include "hip/hip_runtime.h"
// C_main.cu
#include <iostream>
#include <vector>
#include <cstdint>
#include <hip/hip_runtime.h>
#include "C_image.h"     

#include "C_timer.h"
#include "C_render_cpu_baseline.h"
#include "C_render_cpu_threads.h"
// #include "C_render_cpu_openmp.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION      // shouldn't declare if 1_firstP3.cpp were part of RayTracingCUDA.exe under CMakeLists.txt
#include "stb_image_write.h"                // Write jpg


// CUDA kernel for computing gradient on the GPU, saving the Image result as PPM
// Allocates pixel buffer in unified memory via hipMallocManaged
// Launches thousands of threads (1/pixel)
// Practicing GPU acceleration and parallelism via thread heirarchy (grid/block/threadIdx), memory management and synchronization via hipDeviceSynchronize()
    // "embarrassingly parallel" problem can be divided into completely independent sub-problems that can be processed simultaneously with minimal or no communication or dependency between them.
    // "embarrassingly" refers to how easy the parallelization is, as the task is so straightforward that it's almost "embarrassing" to need a complex parallel setup for it.

// Define a device kernel that runs on the GPU and launches many threads
__global__ void gradient_kernel(uint8_t* pixels, int width, int height) {
    // Compute the pixel this GPU thread is responsible for from its block and thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;      // ensure edges of the grid do not overshoot

    // Flip Y like CPU version (top->bottom rendering)
    int jj = (height - 1 - y);

    float r = float(x) / float(width);
    float g = float(jj) / float(height);
    float b = 0.2f;
    
    // Gradient pixel stored as AoS
    int idx = 3 * (y * width + x);
    pixels[idx + 0] = (uint8_t)(255.99f * r);
    pixels[idx + 1] = (uint8_t)(255.99f * g);
    pixels[idx + 2] = (uint8_t)(255.99f * b);
}

int main() {
    const int W = 7680, H = 4320;
    const size_t bytes = W * H * 3;

    // CPU Baseline
    Image img_cpu_base(W, H);   // Image is a CPU-side image class and expects standard CPU memory for saving .ppm and .jpg
    Timer timer_cpu_base;
    timer_cpu_base.tic();

    render_cpu_baseline(img_cpu_base);   // Your CPU single-threaded function
    double cpu_base_time = timer_cpu_base.toc_ms();

    img_cpu_base.write_ppm("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cpu_baseline.ppm");
    stbi_write_jpg("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cpu_baseline.jpg", W, H, 3, img_cpu_base.pixels.data(), 90);
    std::cout << "CPU baseline execution time: " << cpu_base_time << " ms\n";


    // CPU multithreaded
    Image img_cpu_threads(W, H);
    Timer timer_threads;
    timer_threads.tic();

    render_cpu_threads(img_cpu_threads);   // Your CPU multithreaded function
    double cpu_threads_time = timer_threads.toc_ms();

    img_cpu_threads.write_ppm("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cpu_threads.ppm");
    stbi_write_jpg("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cpu_threads.jpg", W, H, 3, img_cpu_threads.pixels.data(), 90);
    std::cout << "CPU multi-threaded time: " << cpu_threads_time << " ms\n";


    // CUDA GPU
    // Unified memory for simplicity
    uint8_t* d_pixels = nullptr;
    hipMallocManaged(&d_pixels, bytes);        // allocate unified memory accessible by both CPU and GPU

    // Set a 2D block size and compute enough blocks to cover the image (ceiling division)
    dim3 block(16, 16);
    dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y);

    // Begin timing
    Timer timer_cuda;
    timer_cuda.tic();

    // Launch the kernel and wait for it to finish
    gradient_kernel << <grid, block >> > (d_pixels, W, H);      // launch kernel
    hipDeviceSynchronize();                                    // wait for kernel to sync

    // End timing
    double cuda_time = timer_cuda.toc_ms();                     // measure elapsed time

    Image img_cuda(W, H);   // copy the unified memory into the CPU Image object and save it
    std::memcpy(img_cuda.pixels.data(), d_pixels, bytes);
    img_cuda.write_ppm("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cuda_output.ppm");
    stbi_write_jpg("C:/Users/ohjin/OneDrive/����/GitHub/RayTracing/RayTracing/cuda_output.jpg", W, H, 3, d_pixels, 90);

    hipFree(d_pixels);     // free GPU memory

    std::cout << "CUDA GPU-accelerated execution time: " << cuda_time << " ms\n";


    // Performance comparison
    std::cout << "GPU speedup vs CPU baseline : " << (cpu_base_time / cuda_time) << "x\n";
    std::cout << "GPU speedup vs CPU threads : " << (cpu_threads_time / cuda_time) << "x\n";
    std::cout << "Multithreading speedup vs baseline : " << (cpu_base_time / cpu_threads_time) << "x\n";


    return 0;
}

// Note: 1200 x 600 = 720,000 pixels; output image is about 33 KB - CPU is bound to be faster than multi-threading or GPU-acceleration
// HD (1920 x 1080 = 2.08M) or 4K (3840 x 2160 = 8.3M) images is where multi-threading & GPU-acceleration would shine
// Benchmark Results (1 trial each):
    // 1200 x 600 or 720,000 pixel image: CPU baseline: 2.842 ms; CPU multi-thread: 7.6332 ms; GPU CUDA: 3.8374 ms
        // GPU speedup vs CPU baseline : 0.7406x; GPU speedup vs CPU threads : 1.9892x; Multithreading speedup vs baseline : 0.37232x
    // 1920 x 1080 or 2,073,600 pixel image: CPU baseline: 7.7165 ms; CPU multi-thread: 9.6203 ms; GPU CUDA: 2.9483 ms
        // GPU speedup vs CPU baseline : 2.61727x; GPU speedup vs CPU threads : 3.263x; Multithreading speedup vs baseline : 0.802106x
    // 3840 x 2160 or 8,294,400 pixel image: CPU baseline: 31.1835 ms; CPU multi-thread: 13.068 ms; GPU CUDA: 3.7394 ms
        // GPU speedup vs CPU baseline : 8.33917x; GPU speedup vs CPU threads : 3.49468x; Multithreading speedup vs baseline : 2.38625x
    // 7680 �� 4320 or 33,177,600 pixel image: CPU baseline: 123.91 ms; CPU multi-thread: 34.9096 ms; GPU CUDA: 3.4855 ms
        // GPU speedup vs CPU baseline : 35.5502x; GPU speedup vs CPU threads : 10.0157x; Multithreading speedup vs baseline : 3.54946x
// Truly shines for 4K and beyond